#include "hip/hip_runtime.h"
#include "backward.cuh"
#include "helpers.cuh"
#include <hip/hip_fp16.h>
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
namespace cg = cooperative_groups;

inline __device__ void warpSum3(float3& val, cg::thread_block_tile<32>& tile){
    val.x = cg::reduce(tile, val.x, cg::plus<float>());
    val.y = cg::reduce(tile, val.y, cg::plus<float>());
    val.z = cg::reduce(tile, val.z, cg::plus<float>());
}

inline __device__ void warpSum2(float2& val, cg::thread_block_tile<32>& tile){
    val.x = cg::reduce(tile, val.x, cg::plus<float>());
    val.y = cg::reduce(tile, val.y, cg::plus<float>());
}

inline __device__ void warpSum(float& val, cg::thread_block_tile<32>& tile){
    val = cg::reduce(tile, val, cg::plus<float>());
}

__global__ void rasterize_backward_kernel(
    const dim3 tile_bounds,
    const dim3 img_size,
    const int32_t* __restrict__ gaussian_ids_sorted,
    const int2* __restrict__ tile_bins,
    const float2* __restrict__ xys,
    const float3* __restrict__ conics,
    const float3* __restrict__ rgbs,
    const int* __restrict__ final_index,
    const float3* __restrict__ v_output,
    float2* __restrict__ v_xy,
    float2* __restrict__ v_xy_abs,
    float3* __restrict__ v_conic,
    float3* __restrict__ v_rgb
) {
    auto block = cg::this_thread_block();
    int32_t tile_id =
        block.group_index().y * tile_bounds.x + block.group_index().x;
    unsigned i =
        block.group_index().y * block.group_dim().y + block.thread_index().y;
    unsigned j =
        block.group_index().x * block.group_dim().x + block.thread_index().x;

    const float px = (float)j + 0.5;
    const float py = (float)i + 0.5;
    // clamp this value to the last pixel
    const int32_t pix_id = min(i * img_size.x + j, img_size.x * img_size.y - 1);

    // keep not rasterizing threads around for reading data
    const bool inside = (i < img_size.y && j < img_size.x);

    // the contribution from gaussians behind the current one
    // float3 buffer = {0.f, 0.f, 0.f};
    // index of last gaussian to contribute to this pixel
    const int bin_final = inside? final_index[pix_id] : 0;

    // have all threads in tile process the same gaussians in batches
    // first collect gaussians between range.x and range.y in batches
    // which gaussians to look through in this tile
    const int2 range = tile_bins[tile_id];
    const int block_size = block.size();
    const int num_batches = (range.y - range.x + block_size - 1) / block_size;

    __shared__ int32_t id_batch[MAX_BLOCK_SIZE];
    __shared__ float2 xy_batch[MAX_BLOCK_SIZE];
    __shared__ float3 conic_batch[MAX_BLOCK_SIZE];
    __shared__ float3 rgbs_batch[MAX_BLOCK_SIZE];

    // df/d_out for this pixel
    const float3 v_out = v_output[pix_id];

    // collect and process batches of gaussians
    // each thread loads one gaussian at a time before rasterizing
    const int tr = block.thread_rank();
    cg::thread_block_tile<32> warp = cg::tiled_partition<32>(block);
    const int warp_bin_final = cg::reduce(warp, bin_final, cg::greater<int>());
    for (int b = 0; b < num_batches; ++b) {
        // resync all threads before writing next batch of shared mem
        block.sync();

        // each thread fetch 1 gaussian from back to front
        // 0 index will be furthest back in batch
        // index of gaussian to load
        // batch end is the index of the last gaussian in the batch
        const int batch_end = range.y - 1 - block_size * b;
        int batch_size = min(block_size, batch_end + 1 - range.x);
        const int idx = batch_end - tr;
        if (idx >= range.x) {
            int32_t g_id = gaussian_ids_sorted[idx];
            id_batch[tr] = g_id;
            xy_batch[tr] = xys[g_id];
            conic_batch[tr] = conics[g_id];
            rgbs_batch[tr] = rgbs[g_id];
        }
        // wait for other threads to collect the gaussians in batch
        block.sync();
        // process gaussians in the current batch for this pixel
        // 0 index is the furthest back gaussian in the batch
        for (int t = max(0,batch_end - warp_bin_final); t < batch_size; ++t) {
            int valid = inside;
            if (batch_end - t > bin_final) {
                valid = 0;
            }
            float alpha;
            float2 delta;
            float3 conic;
            float vis;
            if(valid){
                conic = conic_batch[t];
                float2 xy = xy_batch[t];
                delta = {xy.x - px, xy.y - py};
                float sigma = 0.5f * (conic.x * delta.x * delta.x +
                                            conic.z * delta.y * delta.y) +
                                    conic.y * delta.x * delta.y;
                vis = __expf(-sigma);
                alpha = min(0.99f, vis);
                if (sigma < 0.f || alpha < 1.f / 255.f) {
                    valid = 0;
                }
            }
            // if all threads are inactive in this warp, skip this loop
            if(!warp.any(valid)){
                continue;
            }
            float3 v_rgb_local = {0.f, 0.f, 0.f};
            float3 v_conic_local = {0.f, 0.f, 0.f};
            float2 v_xy_local = {0.f, 0.f};
            float2 v_xy_abs_local = {0.f, 0.f};
            //initialize everything to 0, only set if the lane is valid
            if(valid){

                const float fac = alpha;
                float v_alpha = 0.f;
                v_rgb_local = {fac * v_out.x, fac * v_out.y, fac * v_out.z};

                const float3 rgb = rgbs_batch[t];

                v_alpha += rgb.x * v_out.x;
                v_alpha += rgb.y * v_out.y;
                v_alpha += rgb.z * v_out.z; 

                const float v_sigma = - vis * v_alpha;
                v_conic_local = {0.5f * v_sigma * delta.x * delta.x, 
                                 v_sigma * delta.x * delta.y,
                                 0.5f * v_sigma * delta.y * delta.y};

                v_xy_local = {v_sigma * (conic.x * delta.x + conic.y * delta.y), 
                                    v_sigma * (conic.y * delta.x + conic.z * delta.y)};
                v_xy_abs_local = {abs(v_xy_local.x), abs(v_xy_local.y)};
            }
            warpSum3(v_rgb_local, warp);
            warpSum3(v_conic_local, warp);
            warpSum2(v_xy_local, warp);
            warpSum2(v_xy_abs_local, warp);
            if (warp.thread_rank() == 0) {
                int32_t g = id_batch[t];
                float* v_rgb_ptr = (float*)(v_rgb);
                atomicAdd(v_rgb_ptr + 3*g + 0, v_rgb_local.x);
                atomicAdd(v_rgb_ptr + 3*g + 1, v_rgb_local.y);
                atomicAdd(v_rgb_ptr + 3*g + 2, v_rgb_local.z);
                
                float* v_conic_ptr = (float*)(v_conic);
                atomicAdd(v_conic_ptr + 3*g + 0, v_conic_local.x);
                atomicAdd(v_conic_ptr + 3*g + 1, v_conic_local.y);
                atomicAdd(v_conic_ptr + 3*g + 2, v_conic_local.z);
                
                float* v_xy_ptr = (float*)(v_xy);
                atomicAdd(v_xy_ptr + 2*g + 0, v_xy_local.x);
                atomicAdd(v_xy_ptr + 2*g + 1, v_xy_local.y);

                float* v_xy_abs_ptr = (float*)(v_xy_abs);
                atomicAdd(v_xy_abs_ptr + 2*g + 0, v_xy_abs_local.x);
                atomicAdd(v_xy_abs_ptr + 2*g + 1, v_xy_abs_local.y);
                
            }
        }
    }
}

__global__ void project_gaussians_backward_kernel(
    const int num_points,
    const int* __restrict__ radii,
    const float3* __restrict__ conics,
    const float2* __restrict__ v_xy,
    const float3* __restrict__ v_conic,
    float3* __restrict__ v_cov2d,
    float2* __restrict__ v_mean2d
) {
    unsigned idx = cg::this_grid().thread_rank(); // idx of thread within grid
    if (idx >= num_points || radii[idx] <= 0) {
        return;
    }

    v_mean2d[idx].x = v_xy[idx].x;
    v_mean2d[idx].y = v_xy[idx].y;

    // get v_cov2d
    cov2d_to_conic_vjp(conics[idx], v_conic[idx], v_cov2d[idx]);
}
